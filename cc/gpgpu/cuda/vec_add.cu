
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>


const int N =5;

__global__ void vec_add(float* out, float* a, float* b, int N)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i< N){
		out[i] = a[i] + b[i];
	}
}

