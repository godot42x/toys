#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include "vec_add.cuh"


__global__ void vec_add(float* out, float* a, float* b, int N)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i< N){
		out[i] = a[i] + b[i];
	}
}

