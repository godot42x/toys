
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>


const int N =5;

__global__ void vec_add(float* out, float* a, float* b, int N)
{
	int i = threadIdx.x;
	if(i< N){
		out[i] = a[i] + b[i];
	}
}

int main(){
	float* out, *a ,*b;
	out =  (float*) malloc(sizeof(float)*N);
	a =  (float*) malloc(sizeof(float)*N);
	b =  (float*) malloc(sizeof(float)*N);

	float* d_out, *d_a ,*d_b;
	hipMalloc((void**)&d_out, sizeof(float) * N);
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);

	for(int i =0 ;i< N; ++i){
		a[i] = i;
		b[i]= i+1;
	}

	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);


	vec_add<<<1,N>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);



	for(int i =0 ;i< N; ++i){
		printf("%f ", out[i]);
	}

	hipFree(d_out);
    hipFree(d_a);
    hipFree(d_b);
    free(out);
    free(a);
    free(b);

	return 0;
}
